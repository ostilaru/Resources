#include "cuda.alloc_c1.cuh"

void allocateCudaMemoryC1(int d_batch, int npts, int nstep_valid, int nfft_2x,
                          int do_runabs, int wh_flag,
                          float **d_timesignal,
                          hipComplex **d_spectrum,
                          float **d_segment_timesignal,
                          hipComplex **d_segment_spectrum,
                          float **d_filtered_segment_timesignal,
                          hipComplex **d_filtered_segment_spectrum,
                          hipComplex **d_filter_responses,
                          int filterCount,
                          float **d_weight, float **d_tmp,
                          double **d_sum, double **d_isum,
                          hipfftHandle *planfwd, hipfftHandle *planinv)
{
    // Variables for input and output
    hipMalloc((void **)d_timesignal, d_batch * npts * sizeof(float));
    hipMalloc((void **)d_spectrum, d_batch * nstep_valid * (nfft_2x / 2 + 1) * sizeof(hipComplex));

    // Variables for processing segment data
    hipMalloc((void **)d_segment_timesignal, d_batch * nfft_2x * sizeof(float));
    hipMalloc((void **)d_segment_spectrum, d_batch * nfft_2x * sizeof(hipComplex));
    hipMalloc((void **)d_filter_responses, filterCount * nfft_2x * sizeof(hipComplex));
    hipMalloc((void **)d_sum, d_batch * sizeof(double));
    hipMalloc((void **)d_isum, d_batch * sizeof(double));

    // if whiten, allocate memory for weight and tmp, same size as segment data
    if (!do_runabs && wh_flag)
    {
        hipMalloc((void **)d_weight, d_batch * nfft_2x * sizeof(float));
        hipMalloc((void **)d_tmp, d_batch * nfft_2x * sizeof(float));
    } // if runabs, allocate memory for filterd sac and spec
    else if (do_runabs)
    {
        hipMalloc((void **)d_filtered_segment_timesignal, d_batch * nfft_2x * sizeof(float));
        hipMalloc((void **)d_filtered_segment_spectrum, d_batch * nfft_2x * sizeof(hipComplex));
        hipMalloc((void **)d_weight, d_batch * nfft_2x * sizeof(float));
        hipMalloc((void **)d_tmp, d_batch * nfft_2x * sizeof(float));
    }

    // set up cufft plans
    int rank = 1;
    int n[1] = {nfft_2x};
    int inembed[1] = {nfft_2x};
    int onembed[1] = {nfft_2x};
    int istride = 1;
    int idist = nfft_2x;
    int ostride = 1;
    int odist = nfft_2x;

    hipfftPlanMany(planfwd, 1, n, inembed, istride, idist, onembed,
                  ostride, odist, HIPFFT_R2C, d_batch);
    hipfftPlanMany(planinv, rank, n, inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2R, d_batch);
}

// Free memory for the PathNode linked list
void free_PathList(PathNode *head)
{
    PathNode *current = head;
    PathNode *next_node;

    while (current != NULL)
    {
        next_node = current->next;

        // Free memory for the string
        if (current->path != NULL)
        {
            CpuFree((void **)&current->path);
        }

        // Free memory for the struct itself
        CpuFree((void **)&current);

        current = next_node;
    }
}

void freeMemory(hipfftHandle planfwd, hipfftHandle planinv,
                float **d_timesignal,
                hipComplex **d_spectrum,
                float **d_segment_timesignal,
                hipComplex **d_segment_spectrum,
                float **d_filtered_segment_timesignal,
                hipComplex **d_filtered_segment_spectrum,
                hipComplex **d_filter_responses,
                float **d_weight, float **d_tmp,
                double **d_sum, double **d_isum,
                float **h_timesignal, complex **h_spectrum,
                InOutNodeC1 *pInOutList)
{
    hipfftDestroy(planfwd);
    hipfftDestroy(planinv);

    GpuFree((void **)d_timesignal);
    GpuFree((void **)d_spectrum);
    GpuFree((void **)d_segment_timesignal);
    GpuFree((void **)d_segment_spectrum);
    GpuFree((void **)d_filtered_segment_timesignal);
    GpuFree((void **)d_filtered_segment_spectrum);
    GpuFree((void **)d_filter_responses);

    GpuFree((void **)d_weight);
    GpuFree((void **)d_tmp);

    GpuFree((void **)d_sum);
    GpuFree((void **)d_isum);

    CpuFree((void **)h_timesignal);
    CpuFree((void **)h_spectrum);
    CpuFree((void **)pInOutList);
}
