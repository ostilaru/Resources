#include "hip/hip_runtime.h"
#include "cuda.xc_dual.cuh"
#include "cuda.util.cuh"
#include "segspec.h"
#include <cstddef>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <linux/limits.h>
#include <math.h>
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/sysinfo.h>
#include <sys/time.h>
#include <sys/types.h>
#include <unistd.h>
#include <errno.h>

#define K_LEN_8 8
#define K_LEN_16 16

extern "C"
{
#include "sac.h"
#include "arguproc.h"
#include "read_segspec.h"
#include "read_spec_lst.h"
#include "gen_pair_dual.h"
#include "gen_ccfpath.h"
#include "util.h"
}

pthread_mutex_t g_paramlock = PTHREAD_MUTEX_INITIALIZER;
size_t g_batchload = 0;
size_t g_totalload = 0;

// NOTE: sharedItem
typedef struct
{
  pthread_mutex_t mtx;
  int valid; /* -1: default; 1: ready to file; 2: finish to file */
  char fname[PATH_MAX];
  SACHEAD *phead;
  float *pdata;
} SHAREDITEM;

int create_parent_dir(const char *path)
{
    char *path_copy = strdup(path);
    char *parent_dir = dirname(path_copy);

    if (access(parent_dir, F_OK) == -1)
    {
        create_parent_dir(parent_dir);
        if (mkdir(parent_dir, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH) == -1 && errno != EEXIST)
        {
            free(path_copy);
            return -1;
        }
    }

    free(path_copy);
    return 0;
}

//  pthread_mutex_t g_paramlock;
void *writethrd(void *arg)
{
  struct timespec req, rem;
  req.tv_sec = 0;
  req.tv_nsec = 1000;
  rem.tv_sec = 0;
  rem.tv_nsec = 0;

  size_t writecnt = 0;
  SHAREDITEM *pItem = (SHAREDITEM *)arg;

  size_t batchload = 0;
  size_t totalload = 0;

  pthread_mutex_lock(&g_paramlock);
  batchload = g_batchload;
  totalload = g_totalload;
  pthread_mutex_unlock(&g_paramlock);

  // tag: for debug, check batchload, totalload
  printf("[INFO]: batchload: %ld\n", batchload);
  printf("[INFO]: totalload: %ld\n", totalload);

  while (writecnt < batchload)
  {
    // FIXME: now only write 1 pair
    for (size_t i = 0; i < totalload; i++)
    {
      SHAREDITEM *ptr = pItem + i;
      pthread_mutex_lock(&(ptr->mtx));
      if (ptr->valid == 0)
      {
        // tag: for debug, check ptr->fname
        // printf("[INFO]: ptr->fname: %s\n", ptr->fname);
        // printf("[INFO]: ptr->phead->npts: %d\n", ptr->phead->npts);

        if (write_sac(ptr->fname, *(ptr->phead), ptr->pdata) != 0)
        {
          fprintf(stderr, "ERROR Write output NCF %s error\n", ptr->fname);

          pthread_mutex_unlock(&(ptr->mtx));

          exit(-1);
        }

        ptr->valid = 1;
        writecnt++;
      }
      pthread_mutex_unlock(&(ptr->mtx));
    }
    nanosleep(&req, &rem);
  }
  return NULL;
}

void *writethrd_onlyOnePair(void *arg)
{
  struct timespec req, rem;
  req.tv_sec = 0;
  req.tv_nsec = 1000;
  rem.tv_sec = 0;
  rem.tv_nsec = 0;

  size_t writecnt = 0;
  SHAREDITEM *pItem = (SHAREDITEM *)arg;

  size_t batchload = 0;
  size_t totalload = 0;

  pthread_mutex_lock(&g_paramlock);
  batchload = g_batchload;
  totalload = g_totalload;
  pthread_mutex_unlock(&g_paramlock);

  // tag: for debug, check batchload, totalload
  printf("[INFO]: batchload: %ld\n", batchload);
  printf("[INFO]: totalload: %ld\n", totalload);

  
  for (size_t i = 0; i < 2; i++)
  {
    SHAREDITEM *ptr = pItem + i;
    pthread_mutex_lock(&(ptr->mtx));
    if (ptr->valid == 0)
    {
      // tag: for debug, check ptr->fname
      printf("[INFO]: ptr->fname: %s\n", ptr->fname);
      printf("[INFO]: ptr->phead->npts: %d\n", ptr->phead->npts);

      if (write_sac(ptr->fname, *(ptr->phead), ptr->pdata) != 0)
      {
        fprintf(stderr, "ERROR Write output NCF %s error\n", ptr->fname);

        pthread_mutex_unlock(&(ptr->mtx));

        exit(-1);
      }

      ptr->valid = 1;
      writecnt++;
    }
    pthread_mutex_unlock(&(ptr->mtx));
  }
  nanosleep(&req, &rem);
  
  return NULL;
}

int main(int argc, char **argv)
{
  
  ARGUTYPE argument;
  ArgumentProcess(argc, argv, &argument);
  ARGUTYPE *parg = &argument;

  SPECNODE *pSpecSrcList, *pSpecStaList;
  PAIRNODE *pPairList;

  /* Argumnet parameter */
  float cclength = parg->cclength;
  char *ncf_dir = parg->ncf_dir;
  int gpu_id = parg->gpu_id;
  CUDACHECK(hipSetDevice(gpu_id));

  // Generate list of input src/sta spectrum
  // DONE: rewrite `read_spec_list` to read a station's all year spectrum
  FilePaths *pSrcPaths = read_spec_lst(parg->src_spectrum_lst);
  FilePaths *pStaPaths = read_spec_lst(parg->sta_spectrum_lst);
  
  size_t srccnt = pSrcPaths->count;
  size_t stacnt = pStaPaths->count;

  SEGSPEC spechead;
  read_spechead(pSrcPaths->paths[0], &spechead);
  int nspec = spechead.nspec;
  int nstep = spechead.nstep;
  float delta = spechead.dt;
  int nfft = 2 * (nspec - 1);

  /* get npts of ouput NCF from -cclength to cclength */
  int nhalfcc = (int)floorf(cclength / delta);
  int ncc = 2 * nhalfcc + 1;
  /*********    END OF PRE DEFINING  AND PARSING    ***********/

  /* Alloc static CPU memory */
  complex *src_buffer = NULL; // input src spectrum
  complex *sta_buffer = NULL; // input sta spectrum
  float *ncf_buffer = NULL;   // output ncf data

  size_t total_cnt = 0;
  total_cnt = srccnt + stacnt;  // NOTE: total_cnt means total number of spectrum files

  size_t vec_cnt = nstep * nspec;              // number of point in a spectrum file
  size_t vec_size = vec_cnt * sizeof(complex); // size of a spectrum file

  // TODO: here xc num is n*n, but now only 2 station, so xc num is n
  /*
  // least size of CPU memory required
  size_t fixedCpuRam = total_cnt * vec_size                  // spectrum data buffer
                       + total_cnt * sizeof(SPECNODE)        // spectrum node
                       + srccnt * stacnt * sizeof(PAIRNODE); // pair node
  */
  // least size of CPU memory required
  size_t fixedCpuRam = total_cnt * vec_size                  // spectrum data buffer
                       + total_cnt * sizeof(SPECNODE)        // spectrum node
                       + std::min(srccnt, stacnt) * sizeof(PAIRNODE); // pair node

  // tag: check fixedCpuRam for debug
  printf("[INFO]: fixedCpuRam: %.3f GB\n", (float)fixedCpuRam / (1024 * 1024 * 1024));

  /* The unitCpuram represent the memory used to write out file */
  size_t unitCpuram = nfft * sizeof(float) + sizeof(SHAREDITEM);

  // tag: check unitCpuram for debug
  printf("[INFO]: unitCpuram: %.3f MB\n", (float)unitCpuram / (1024 * 1024));

  size_t h_batch = EstimateCpuBatch(fixedCpuRam, unitCpuram);

  // tag: check h_batch for debug
  printf("[INFO]: h_batch: %ld\n", h_batch);

  // allocate CPU memory for spectrum node and pair node
  CpuMalloc((void **)&pSpecSrcList,
            srccnt * sizeof(SPECNODE)); // src spectrum node
  CpuMalloc((void **)&pSpecStaList,
            stacnt * sizeof(SPECNODE)); // sta spectrum node
  // TODO: here xc num is n*n, but now only 2 station, so xc num is n
  /*
  CpuMalloc((void **)&pPairList,
            srccnt * stacnt * sizeof(PAIRNODE)); // pair node
  */
  CpuMalloc((void **)&pPairList,
            std::min(srccnt, stacnt) * sizeof(PAIRNODE)); // pair node
  
  // Allocate CPU memory for spectrum data buffer
  CpuMalloc((void **)&src_buffer, srccnt * vec_size); // src spectrum data buffer
  CpuMalloc((void **)&sta_buffer, stacnt * vec_size);

  // init src spectrum node, mapping .pdata point to data Buffer
  for (size_t i = 0; i < srccnt; i++)
  {
    pSpecSrcList[i].pdata = src_buffer + i * nstep * nspec;
  }
 
  for (size_t i = 0; i < stacnt; i++)
  {
    pSpecStaList[i].pdata = sta_buffer + i * nstep * nspec;
  }
  // reading data from filenode_list to speclist.pdata
  // spec.pdata has already been mapped to srcBuffer/staBuffer

  // tag: for debug, check if run here
  printf("[INFO]: already init src spectrum node!\n");

  GenSpecArray(pSrcPaths, pSpecSrcList);
  GenSpecArray(pStaPaths, pSpecStaList);

  // tag: for debug, check if run here
  printf("[INFO]: already finish GenSpecArray()!\n");

  // DONE: GeneratePair_dual() need to be fixed, add filenameDate cmp
  size_t paircnt = GeneratePair_dual(pPairList, pSpecSrcList, srccnt, pSpecStaList, stacnt);

  // tag: for debug, check paircnt
  printf("[INFO]: paircnt: %ld\n", paircnt);

  h_batch = (h_batch > paircnt) ? paircnt : h_batch;
  
  /* Alloc cpu dynamic memory */
  CpuMalloc((void **)&ncf_buffer, h_batch * nfft * sizeof(float));

  // Set the head of output NCF of each pair src file and sta file
  for (size_t i = 0; i < paircnt; i++)
  {
    SACHEAD *phd_ncf = &(pPairList[i].headncf);
    SEGSPEC *phd_src = &(pSpecSrcList[pPairList[i].srcidx].head);
    SEGSPEC *phd_sta = &(pSpecStaList[pPairList[i].staidx].head);
    SacheadProcess(phd_ncf, phd_src, phd_sta, delta, ncc, cclength);
  }

  /* Slave thread  property */
  SHAREDITEM *pItem;
  CpuMalloc((void **)&pItem, paircnt * sizeof(SHAREDITEM));
  for (size_t i = 0; i < paircnt; i++)
  {
    SHAREDITEM *ptr = pItem + i;
    pthread_mutex_init(&ptr->mtx, NULL);
    pthread_mutex_lock(&ptr->mtx);
    ptr->valid = -1;
    pthread_mutex_unlock(&ptr->mtx);
  }

  // TODO: now we need stack process after xc, so we need more CPU memory
  // ---------------------------stack memory-------------------------------------------
  SACHEAD template_hd = sac_null, infilehd = sac_null;

  size_t nstack = 0;
  size_t k = 0;

  size_t ncf_num = paircnt;
  template_hd = pPairList[0].headncf;

  char *ncf_filepath = GetNcfPath(pSpecSrcList[(pPairList + 0)->srcidx].filepath,
                     pSpecStaList[(pPairList + 0)->staidx].filepath,
                     ncf_dir);

  char template_path[256];
  strcpy(template_path, ncf_filepath);
  char *base_name = basename(template_path); 

  /* Extract the required fields */
  char *fields[5];
  int i = 0;
  char *token = strtok(base_name, ".");
  while (token != NULL)
  {
    fields[i++] = token;
    token = strtok(NULL, ".");
  }

  // NOTE: filename's 1st part is sta-pair, 2nd part is component-pair
  // example: AAKH-ABNH.U-U.sac
  char *sta_pair = fields[0];
  char *component_pair = fields[1];

  char *rest = sta_pair;
  char *saveptr;

  token = strtok_r(rest, "-", &saveptr);
  char *kevnm = strtok(sta_pair, "-");
  rest = NULL;
  char *kstnm = strtok_r(rest, "-", &saveptr);

  // tag: for debug, check for npts, ncf_num, ncf_filepath, kevnm, kstnm = AAKH, ABNH, component_pair = U-U
  printf("[INFO]: template_hd.npts: %d\n", template_hd.npts);
  printf("[INFO]: ncf_num: %ld\n", ncf_num);
  printf("[INFO]: ncf_filepath: %s\n", ncf_filepath);
  printf("[INFO]: kevnm: %s, kstnm: %s\n", kevnm, kstnm);
  printf("[INFO]: component_pair: %s\n", component_pair);

  /* Write fields to the sac header */
  strncpy(template_hd.kstnm, kstnm, K_LEN_8);
  strncpy(template_hd.kevnm, kevnm, K_LEN_16);
  strncpy(template_hd.kcmpnm, component_pair, K_LEN_8);

  int npts = template_hd.npts;
  SACHEAD hdstack = template_hd;

  /* change the reference time nzyear nzjday nzhour nzmin nzsec nzmsec */
  hdstack.nzyear = 2010;
  hdstack.nzjday = 214;
  hdstack.nzhour = 16;
  hdstack.nzmin = 0;
  hdstack.nzsec = 0;
  hdstack.nzmsec = 0;

  /* Copy coordinate infomation from first sac file */
  hdstack.stla = template_hd.stla;
  hdstack.stlo = template_hd.stlo;
  hdstack.evla = template_hd.evla;
  hdstack.evlo = template_hd.evlo;

  hdstack.dist = template_hd.dist;
  hdstack.az = template_hd.az;
  hdstack.baz = template_hd.baz;
  hdstack.gcarc = template_hd.gcarc;

  float *stackcc = NULL;
  stackcc = (float *)malloc(sizeof(float) * npts);
  nstack = 0;

  // set stackcc to zero
  for (k = 0; k < npts; k++)
  {
    stackcc[k] = 0.0;
  }

  // ---------------------------stack memory-------------------------------------------
  
  /* Alloc gpu static memory */
  // cufft handle
  hipfftHandle plan;
  int rank = 1;
  int n[1] = {nfft};
  int inembed[1] = {nfft};
  int onembed[1] = {nfft};
  int istride = 1;
  int idist = nfft;
  int ostride = 1;
  int odist = nfft;
  hipfftType type = HIPFFT_C2R;
  int numType = 1;
  hipfftType typeArr[1] = {type};

  size_t unitgpuram = sizeof(PAIRNODE)               // input pair node
                      + 2 * nfft * sizeof(complex)   // input src spectrum
                      + 2 * nfft * sizeof(float);    // output ncf data
  size_t fixedGpuRam = total_cnt * vec_size;

  // tag: for debug, check vec_size, unitgpuram, fixedGpuRam
  printf("[INFO]: -----------------------------GPU Alloc Start-----------------------------------------\n");
  printf("[INFO]: total_cnt: %ld\n", total_cnt);
  printf("[INFO]: vec_size: %.3f MB\n", (float)vec_size / (1024 * 1024));
  printf("[INFO]: unitgpuram: %.3f MB\n", (float)unitgpuram / (1024 * 1024));
  printf("[INFO]: fixedGpuRam: %.3f GB\n", (float)fixedGpuRam / (1024 * 1024 * 1024));

  // Estimate the maximum number of batch
  size_t d_batch = EstimateGpuBatch(gpu_id, fixedGpuRam, unitgpuram, numType,
                                    rank, n, inembed, istride, idist, onembed,
                                    ostride, odist, typeArr);
  // set the maximum number of batch
  d_batch = (d_batch > h_batch) ? h_batch : d_batch;

  // tag: for debug, check d_batch
  printf("[INFO]: d_batch: %ld\n", d_batch);

  // Define GPU memory pointer
  hipComplex *d_src_spectrum = NULL;         // input src spectrum
  hipComplex *d_sta_spectrum = NULL;         // input sta spectrum
  hipComplex *d_segment_ncf_spectrum = NULL; // output ncf data, segment in spectrum
  hipComplex *d_total_ncf_spectrum = NULL;   // output ncf data, total in spectrum
  float *d_total_ncf = NULL;                // output ncf data, time signal
  PAIRNODE *d_pairlist = NULL;              // pair node

  // Allocate GPU memory for spectrum node data buffer for input
  GpuMalloc((void **)&d_src_spectrum, srccnt * vec_size);
  GpuMalloc((void **)&d_sta_spectrum, stacnt * vec_size);

  // Copy spectrum data from CPU buffer to GPU
  CUDACHECK(hipMemcpy(d_src_spectrum, src_buffer, srccnt * vec_size, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_sta_spectrum, sta_buffer, stacnt * vec_size, hipMemcpyHostToDevice));

  /* Alloc gpu dynamic memory with d_batch */
  CufftPlanAlloc(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, type, d_batch);

  GpuMalloc((void **)&d_pairlist, d_batch * sizeof(PAIRNODE));
  GpuMalloc((void **)&d_segment_ncf_spectrum, d_batch * nfft * sizeof(complex));
  GpuMalloc((void **)&d_total_ncf_spectrum, d_batch * nfft * sizeof(complex));
  GpuMalloc((void **)&d_total_ncf, d_batch * nfft * sizeof(float));

  size_t globalidx = 0;

  // tag: starttime
  struct timespec start_time, end_time;
  clock_gettime(CLOCK_MONOTONIC, &start_time);

  printf("[INFO]: Doing Cross Correlation!\n");
  for (size_t h_finishcnt = 0; h_finishcnt < paircnt; h_finishcnt += h_batch)
  {
    // Set the number of [h_proccnt]: how many ncfs will be written to disk
    size_t h_proccnt = (h_finishcnt + h_batch > paircnt) ? (paircnt - h_finishcnt) : h_batch;

    // tag: for debug, check h_proccnt
    printf("[INFO]: h_proccnt: %ld\n", h_proccnt);

    // Set the memory of [ncfBuffer] to zero
    memset(ncf_buffer, 0, h_batch * nfft * sizeof(float));

    // tag: for debug, check ncf_buffer
    printf("[INFO]: ncf_buffer: %.3f MB\n", (float)h_batch * nfft * sizeof(float) / (1024 * 1024));

    pthread_mutex_lock(&g_paramlock);   // lock
    g_totalload = paircnt;              // total number of pairs
    g_batchload = h_proccnt;            // number of pairs in this batch
    pthread_mutex_unlock(&g_paramlock); // unlock

    // FIXME: here create a new thread for write sac file
    pthread_t tid;
    pthread_create(&tid, NULL, writethrd, (void *)pItem);

    // Launch GPU processing
    for (size_t d_finishcnt = 0; d_finishcnt < h_proccnt; d_finishcnt += d_batch)
    {
      hipMemset(d_total_ncf, 0, d_batch * nfft * sizeof(float));

      size_t d_proccnt = (d_finishcnt + d_batch > h_proccnt) ? (h_proccnt - d_finishcnt) : d_batch;

      // tag: for debug, check for d_proccnt = 10,10,...,10,5
      // printf("[INFO]: d_proccnt: %ld\n", d_proccnt);

      CUDACHECK(hipMemcpy(d_pairlist, pPairList + h_finishcnt + d_finishcnt,
                           d_proccnt * sizeof(PAIRNODE),
                           hipMemcpyHostToDevice));

      CUDACHECK(hipMemset(d_total_ncf_spectrum, 0, d_proccnt * nfft * sizeof(hipComplex)));
      dim3 dimgrd, dimblk;
      DimCompute(&dimgrd, &dimblk, nspec, d_proccnt);
      // NOTE: process each step, example: divide 24h into 12 steps
      for (size_t stepidx = 0; stepidx < nstep; stepidx++)
      {
        /* step by step cc */
        /* Reset temp ncf to zero */
        CUDACHECK(hipMemset(d_segment_ncf_spectrum, 0, d_proccnt * nfft * sizeof(hipComplex)));

        // TODO: `cmuldual2DKernel` need to be rewrite
        cmuldual2DKernel<<<dimgrd, dimblk>>>(d_src_spectrum, vec_cnt, stepidx * nspec,
                                             d_sta_spectrum, vec_cnt, stepidx * nspec,
                                             d_pairlist, d_proccnt, d_segment_ncf_spectrum, nfft, nspec);

        // TODO: `csum2DKernel` need to be rewrite                                     
        csum2DKernel<<<dimgrd, dimblk>>>(d_total_ncf_spectrum, nfft, d_segment_ncf_spectrum, nfft, nspec, d_proccnt, nstep);
      }
      hipfftExecC2R(plan, (hipfftComplex *)d_total_ncf_spectrum, (hipfftReal *)d_total_ncf);
      DimCompute(&dimgrd, &dimblk, nfft, d_proccnt);
      InvNormalize2DKernel<<<dimgrd, dimblk>>>(d_total_ncf, nfft, nfft, d_proccnt, delta);
      hipMemcpy(ncf_buffer + d_finishcnt * nfft, d_total_ncf, d_proccnt * nfft * sizeof(float), hipMemcpyDeviceToHost);

      // FIXME: here cuda_calc finished, start to stack
      
      for (size_t i = 0; i < d_proccnt; i++)
      {
        SHAREDITEM *ptr = pItem + globalidx;
        pthread_mutex_lock(&(ptr->mtx));
        if (ptr->valid == -1)
        {
          GenCCFPath(ptr->fname,
                     pSpecSrcList[(pPairList + globalidx)->srcidx].filepath,
                     pSpecStaList[(pPairList + globalidx)->staidx].filepath,
                     ncf_dir);

          ptr->phead = &((pPairList + globalidx)->headncf);
          ptr->pdata = ncf_buffer + (d_finishcnt + i) * nfft + nspec - nhalfcc - 1;
          ptr->valid = 0;
        }
        pthread_mutex_unlock(&(ptr->mtx));
        globalidx++;
      }

    }

    pthread_join(tid, NULL);
  }

  // TODO: stack process
  // -----------------------------------------------------------------------------
  for (size_t i = 0; i < ncf_num; i++) {
    for (k = 0; k < npts; k++) {
      stackcc[k] = stackcc[k] + pItem[i].pdata[k];
    }
    nstack++;
  }

  // tag: for debug, check stackcc
  int normalize = 1;

  if (normalize == 1)
  {
    for (k = 0; k < npts; k++)
    {
      stackcc[k] /= ncf_num;
    }
  }

  hdstack.unused27 = nstack;
  char *out_sac = "/home/woodwood/hpc/station_2/ncf_hinet_AAKH_ABNH/stack/AAKH-ABNH/AAKH-ABNH.U-U.sac";
  char *out_sac_copy = strdup(out_sac);
  if (create_parent_dir(out_sac) == -1)
  {
    fprintf(stderr, "Error creating directory %s: ", dirname(out_sac_copy));
    perror(NULL);
    free(out_sac_copy);
    return 1;
  }
  write_sac(out_sac, hdstack, stackcc);
  // -----------------------------------------------------------------------------

  // tag: endtime
  clock_gettime(CLOCK_MONOTONIC, &end_time);
  double elapsed_time = (end_time.tv_sec - start_time.tv_sec) +
                      (end_time.tv_nsec - start_time.tv_nsec) / 1e9;
  printf("[INFO]: Elapsed time: %.6f seconds\n", elapsed_time);


  /* Free cpu memory */

  for (size_t i = 0; i < paircnt; i++)
  {
    pthread_mutex_destroy(&((pItem + i)->mtx));
  }

  printf("[INFO]: Finish Cross Correlation!\n");

  free(stackcc);

  CpuFree((void **)&pItem);

  CpuFree((void **)&src_buffer);
  CpuFree((void **)&sta_buffer);
  CpuFree((void **)&ncf_buffer);

  CpuFree((void **)&pSpecSrcList);
  CpuFree((void **)&pSpecStaList);
  CpuFree((void **)&pPairList);

  // Free gpu memory
  GpuFree((void **)&d_src_spectrum);
  GpuFree((void **)&d_sta_spectrum);
  GpuFree((void **)&d_segment_ncf_spectrum);
  GpuFree((void **)&d_total_ncf_spectrum);
  GpuFree((void **)&d_total_ncf);

  CUFFTCHECK(hipfftDestroy(plan));
  freeFilePaths(pSrcPaths);
  freeFilePaths(pStaPaths);

  return 0;
}
