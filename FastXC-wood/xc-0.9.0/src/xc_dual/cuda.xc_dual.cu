#include "hip/hip_runtime.h"
#include "cuda.xc_dual.cuh"

// TODO: this function need to be rewrite
__global__ void cmuldual2DKernel(hipComplex *d_specsrcvec, size_t srcpitch, size_t srcoffset,
                                 hipComplex *d_specstavec, size_t stapitch, size_t staoffset,
                                 PAIRNODE *d_pairlist, size_t paircnt,
                                 hipComplex *d_segncfvec, size_t ncfpitch,
                                 int nspec)
{
  // get the index of the current thread
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;

  // check if the index is out of bound
  if (col < nspec && row < paircnt)
  {
    // NOTE: pitch is used for the 2D array, offset is used for the 1D array
    size_t idx = row * ncfpitch + col;

    // get the index of the source and station
    size_t srcrow, starow;
    size_t srcidx, staidx;
    srcrow = d_pairlist[row].srcidx;
    starow = d_pairlist[row].staidx;
    srcidx = srcrow * srcpitch + srcoffset + col;
    staidx = starow * stapitch + staoffset + col;

    // hipComplex src = d_specsrcvec[srcidx];
    // hipComplex sta_conj =
    //     make_hipComplex(d_specstavec[staidx].x, -d_specstavec[staidx].y);
        
    hipComplex sta = d_specstavec[staidx];
    hipComplex src_conj =
        make_hipComplex(d_specsrcvec[srcidx].x, -d_specsrcvec[srcidx].y);

    if (col == 0)
    {
      d_segncfvec[idx] = make_hipComplex(0, 0);
    }
    else
    {
      // hipComplex mul_result = hipCmulf(src, sta_conj);
      hipComplex mul_result = hipCmulf(src_conj, sta);
      int sign = (col % 2 == 0) ? 1 : -1;
      d_segncfvec[idx].x = sign * mul_result.x;
      d_segncfvec[idx].y = sign * mul_result.y;
    }
  }
}

// sum2dKernel is used to sum the 2D array of float, not used in the current version
__global__ void sum2DKernel(float *d_finalccvec, int dpitch, float *d_segncfvec,
                            int spitch, size_t width, size_t height,
                            int nstep)
{
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int sidx = row * spitch + col;
    int didx = row * dpitch + col;
    d_finalccvec[didx] += (d_segncfvec[sidx] / nstep);
  }
}

// TODO: this function need to be rewrite
__global__ void csum2DKernel(hipComplex *d_total_spectrum, int dpitch,
                             hipComplex *d_segment_spectrum, int spitch,
                             size_t width, size_t height, int nstep)
{
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int sidx = row * spitch + col;
    int didx = row * dpitch + col;
    hipComplex temp = d_segment_spectrum[sidx];
    temp.x /= nstep; // divide the real part by nstep
    temp.y /= nstep; // divide the imaginary part by nstep

    d_total_spectrum[didx] = hipCaddf(d_total_spectrum[didx], temp);
  }
}

__global__ void InvNormalize2DKernel(float *d_segdata, size_t pitch,
                                     size_t width, size_t height, float dt)
{
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  double weight = 1.0 / (width * dt);
  if (row < height && col < width)
  {
    size_t idx = row * pitch + col;
    d_segdata[idx] *= weight;
  }
}
