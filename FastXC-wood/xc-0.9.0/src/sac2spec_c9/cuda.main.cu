/* last updated by wangjx@20230715 */

#include "cuda.alloc_c9.cuh"
#include "cuda.whiten_c9.cuh"
#include "cuda.runabs_c9.cuh"
#include "cuda.onebit.cuh"
#include "cuda.estimatebatch_c9.cuh"
#include "cuda.fft_normalize.cuh"
#include "cuda.preprocess.cuh"
#include "cuda.util.cuh"

extern "C"
{
#include "design_filter_response.h"
#include "arguproc.h"
#include "in_out_node_c9.h"
#include "path_node.h"
#include "cal_nseg.h"
#include "read_filelist.h"
#include "find_whiten_flag.h"
#include "par_read_sac_c9.h"
#include "par_write_spec_c9.h"
#include "sac.h"
#include "util.h"
}

int main(int argc, char **argv)
{
    // Parsing arguments
    ARGUTYPE argument;
    ArgumentProcess(argc, argv, &argument);

    // read in sac path list file
    PathNode *pInFileList_1 = readPathList(argument.sacin_lst_1);
    PathNode *pInFileList_2 = readPathList(argument.sacin_lst_2);
    PathNode *pInFileList_3 = readPathList(argument.sacin_lst_3);

    // read in spec output path list file
    PathNode *pOutFileList_1 = readPathList(argument.specout_lst_1);
    PathNode *pOutFileList_2 = readPathList(argument.specout_lst_2);
    PathNode *pOutFileList_3 = readPathList(argument.specout_lst_3);

    // create output_dir for .segspec files
    createDirectories(pOutFileList_1);
    createDirectories(pOutFileList_2);
    createDirectories(pOutFileList_3);

    // Turn file chain input list chain into array
    FilePathArray InPaths_1 = PathList2Array(pInFileList_1);
    FilePathArray InPaths_2 = PathList2Array(pInFileList_2);
    FilePathArray InPaths_3 = PathList2Array(pInFileList_3);

    FilePathArray OutPaths_1 = PathList2Array(pOutFileList_1);
    FilePathArray OutPaths_2 = PathList2Array(pOutFileList_2);
    FilePathArray OutPaths_3 = PathList2Array(pOutFileList_3);

    // Parse whiten and normalization parameters
    int wh_before = 0, wh_after = 0, do_runabs = 0, do_onebit = 0;
    find_whiten_flag(argument.whitenType, argument.normalizeType, &wh_before, &wh_after, &do_runabs, &do_onebit);

    // Parse frequcny band
    float freq_low_limit = argument.freq_low_limit;
    float freq_high_limit = argument.freq_high_limit;

    // count the number of input files
    size_t nValid = InPaths_1.count;

    // set the gpu_id
    int gpu_id = argument.gpu_id;
    hipSetDevice(gpu_id);

    // read npts and delta form the file file of pInFileList
    SACHEAD sachd;
    if (read_sachead(pInFileList_1->path, &sachd) != 0)
    {
        fprintf(stderr, "ERROR reading first SACHEAD\n");
        exit(EXIT_FAILURE);
    }
    int npts = sachd.npts;
    float delta = sachd.delta;

    // calculate the nseg
    int nseg = cal_nseg(argument.seglen, npts, delta);
    int nstep = npts / nseg;

    // create nfft_2x for zero padding data
    int nfft_2x = nseg * 2;
    int nspec = nfft_2x / 2 + 1;
    float df = 1.0 / (nfft_2x * delta);

    // Parse the option skip_step
    int skip_step = argument.skip_step;
    int nstep_valid = nstep;
    if (skip_step != -1 && skip_step < nstep)
    {
        nstep_valid = nstep - 1;
        printf("We will skip [no. %d] step\n", skip_step);
    }

    // read in filter file
    int filter_count = 0;
    ButterworthFilter *filter = readButterworthFilters(argument.filter_file, &filter_count);
    printButterworthFilters(filter, filter_count);
    FilterResp *myResp = processButterworthFilters(filter, filter_count, df, nspec);

    // print the first filter responses
    // printf("First Filter Responses:\n");
    // printf("freq_low = %f\n", myResp[0].freq_low);
    // for (int i = 0; i < nspec; i++)
    // {
    //     printf("%f %f\n", myResp[0].response[i].x, myResp[0].response[i].y);
    // }
    // exit(0);

    // set smooth npts for whitenning, modified from yao's code 0.02/df
    int nsmooth = int(0.02 * nfft_2x * delta * 2);
    nsmooth = (nsmooth > 22) ? nsmooth : 11;

    // ********* Calculate CPU memory *********** //
    float *h_timesignal_1 = NULL;
    float *h_timesignal_2 = NULL;
    float *h_timesignal_3 = NULL;

    complex *h_spectrum_1 = NULL;
    complex *h_spectrum_2 = NULL;
    complex *h_spectrum_3 = NULL;

    InOutNodeC9 *pInOutList = NULL;

    size_t unit_timesignal_size = 3 * npts * sizeof(float);                // input sac data
    size_t unit_spectrum_size = 3 * nstep_valid * nspec * sizeof(complex); // output total spectrum
    size_t unit_InOutNode_size = sizeof(InOutNodeC9);                      // contain head/path/data
    size_t unit_thread_write_size = sizeof(thread_info_write);             // contain thread_info_write
    size_t unit_thread_read_size = sizeof(thread_info_read);               // contain thread_info_read

    size_t unitCpuRam = unit_timesignal_size +
                        unit_spectrum_size +
                        unit_InOutNode_size +
                        unit_thread_write_size +
                        unit_thread_read_size;

    size_t h_batch = EstimateCpuBatch(unitCpuRam, argument.thread_num);
    h_batch = (h_batch > nValid) ? nValid : h_batch;

    size_t wh_flag = wh_after || wh_before;

    size_t d_batch = EstimateGpuBatchC9(gpu_id, npts, nfft_2x, nstep,
                                        wh_flag, do_runabs);

    d_batch = (d_batch > h_batch) ? h_batch : d_batch;
    h_batch = d_batch;

    printf("[GPU no.%d]: cpu_batch = %ld\n", gpu_id, h_batch);
    printf("[GPU no.%d]: gpu_batch = %ld\n", gpu_id, d_batch);

    // Allocate CPU memory
    CpuMalloc((void **)&pInOutList, h_batch * unit_InOutNode_size);

    CpuMalloc((void **)&h_timesignal_1, h_batch * unit_timesignal_size);
    CpuMalloc((void **)&h_spectrum_1, h_batch * unit_spectrum_size);

    CpuMalloc((void **)&h_timesignal_2, h_batch * unit_timesignal_size);
    CpuMalloc((void **)&h_spectrum_2, h_batch * unit_spectrum_size);

    CpuMalloc((void **)&h_timesignal_3, h_batch * unit_timesignal_size);
    CpuMalloc((void **)&h_spectrum_3, h_batch * unit_spectrum_size);

    // Initializing the memory for InOutNode
    for (size_t i = 0; i < h_batch; i++)
    {
        size_t sacpathSize = MAXPATH * sizeof(char);
        size_t specpathSize = MAXPATH * sizeof(char);
        size_t spechdSize = sizeof(SEGSPEC);
        size_t sachdSize = sizeof(SACHEAD);
        CpuMalloc((void **)&(pInOutList[i].sacpath_1), sacpathSize);
        CpuMalloc((void **)&(pInOutList[i].sacpath_2), sacpathSize);
        CpuMalloc((void **)&(pInOutList[i].sacpath_3), sacpathSize);

        CpuMalloc((void **)&(pInOutList[i].specpath_1), specpathSize);
        CpuMalloc((void **)&(pInOutList[i].specpath_2), specpathSize);
        CpuMalloc((void **)&(pInOutList[i].specpath_3), specpathSize);

        CpuMalloc((void **)&(pInOutList[i].sac_hd), sachdSize);
        CpuMalloc((void **)&(pInOutList[i].segspec_hd), spechdSize);

        pInOutList[i].timesignal_1 = h_timesignal_1 + i * npts;
        pInOutList[i].spectrum_1 = h_spectrum_1 + i * nstep_valid * nspec;

        pInOutList[i].timesignal_2 = h_timesignal_2 + i * npts;
        pInOutList[i].spectrum_2 = h_spectrum_2 + i * nstep_valid * nspec;

        pInOutList[i].timesignal_3 = h_timesignal_3 + i * npts;
        pInOutList[i].spectrum_3 = h_spectrum_3 + i * nstep_valid * nspec;

        pInOutList[i].nspec = nspec;
        pInOutList[i].nstep = nstep_valid;
        pInOutList[i].df = df;
        pInOutList[i].dt = delta;
    }

    // ********** Allocate memory for GPU ************ //

    // Initialize the memory for GPU
    float *d_timesignal_1 = NULL;
    float *d_timesignal_2 = NULL;
    float *d_timesignal_3 = NULL;

    hipComplex *d_spectrum_1 = NULL;
    hipComplex *d_spectrum_2 = NULL;
    hipComplex *d_spectrum_3 = NULL;

    float *d_segment_timesignal_1 = NULL;
    float *d_segment_timesignal_2 = NULL;
    float *d_segment_timesignal_3 = NULL;

    hipComplex *d_segment_spectrum_1 = NULL;
    hipComplex *d_segment_spectrum_2 = NULL;
    hipComplex *d_segment_spectrum_3 = NULL;

    float *d_filtered_segment_timesignal_1 = NULL;
    float *d_filtered_segment_timesignal_2 = NULL;
    float *d_filtered_segment_timesignal_3 = NULL;

    hipComplex *d_filtered_segment_spectrum_1 = NULL;
    hipComplex *d_filtered_segment_spectrum_2 = NULL;
    hipComplex *d_filtered_segment_spectrum_3 = NULL;

    hipComplex *d_filter_responses = NULL;

    float *d_weight_1 = NULL; // weight for each segment spectrum data
    float *d_weight_2 = NULL;
    float *d_weight_3 = NULL;

    float *d_tmp = NULL;   // used in npsmooth
    double *d_sum = NULL;  // used in rtr and rdc
    double *d_isum = NULL; // used in rtr and rdc

    hipfftHandle planfwd; // forward fft
    hipfftHandle planinv; // inverse fft

    // Allocating memory in GPU Device
    allocateCudaMemoryC9(d_batch, npts, nstep_valid, nfft_2x,
                         do_runabs, wh_flag,
                         &d_timesignal_1, &d_spectrum_1,
                         &d_timesignal_2, &d_spectrum_2,
                         &d_timesignal_3, &d_spectrum_3,
                         &d_segment_timesignal_1, &d_segment_spectrum_1,
                         &d_segment_timesignal_2, &d_segment_spectrum_2,
                         &d_segment_timesignal_3, &d_segment_spectrum_3,
                         &d_filtered_segment_timesignal_1, &d_filtered_segment_spectrum_1,
                         &d_filtered_segment_timesignal_2, &d_filtered_segment_spectrum_2,
                         &d_filtered_segment_timesignal_3, &d_filtered_segment_spectrum_3,
                         &d_filter_responses,
                         filter_count,
                         &d_weight_1, &d_weight_2, &d_weight_3,
                         &d_tmp, &d_sum, &d_isum,
                         &planfwd, &planinv);

    // ********** MAIN PROCESSING PART ********** /

    // copy each filter response to GPU and store freq_low array
    float freq_lows[filter_count];
    for (int i = 0; i < filter_count; i++)
    {
        freq_lows[i] = myResp[i].freq_low;
        CUDACHECK(hipMemcpy2D(d_filter_responses, nfft_2x * sizeof(hipComplex), myResp[i].response, nfft_2x * sizeof(hipComplex),
                               nfft_2x * sizeof(hipComplex), 1, hipMemcpyHostToDevice));
    }

    for (size_t h_finishcnt = 0; h_finishcnt < nValid; h_finishcnt += h_batch)
    {
        size_t h_proccnt = (h_finishcnt + h_batch > nValid) ? nValid - h_finishcnt : h_batch;

        // Setup Input Output Chain
        for (size_t i = h_finishcnt, j = 0; i < h_finishcnt + h_proccnt; i++, j++)
        {
            pInOutList[j].sacpath_1 = InPaths_1.paths[i];
            pInOutList[j].specpath_1 = OutPaths_1.paths[i];

            pInOutList[j].sacpath_2 = InPaths_2.paths[i];
            pInOutList[j].specpath_2 = OutPaths_2.paths[i];

            pInOutList[j].sacpath_3 = InPaths_3.paths[i];
            pInOutList[j].specpath_3 = OutPaths_3.paths[i];
        }

        ThreadPoolRead *read_pool = create_threadpool_read(MAX_THREADS);
        ThreadPoolWrite *write_pool = create_threadpool_write(MAX_THREADS);

        // Set [h_timesignal_*] and [h_spectrum_*] to zero
        memset(h_timesignal_1, 0, h_proccnt * npts * sizeof(float));
        memset(h_spectrum_1, 0, h_proccnt * nstep_valid * nspec * sizeof(complex));

        memset(h_timesignal_2, 0, h_proccnt * npts * sizeof(float));
        memset(h_spectrum_2, 0, h_proccnt * nstep_valid * nspec * sizeof(complex));

        memset(h_timesignal_3, 0, h_proccnt * npts * sizeof(float));
        memset(h_spectrum_3, 0, h_proccnt * nstep_valid * nspec * sizeof(complex));

        parallel_read_sac_c9(read_pool, h_proccnt, pInOutList, MAX_THREADS);
        // Launch GPU Processing Part

        dim3 dimgrd, dimblk;
        for (size_t d_finishcnt = 0; d_finishcnt < h_proccnt; d_finishcnt += d_batch)
        {
            size_t d_proccnt = (d_finishcnt + d_batch > h_proccnt) ? h_proccnt - d_finishcnt : d_batch;

            CUDACHECK(hipMemset(d_timesignal_1, 0, d_proccnt * npts * sizeof(float)));
            CUDACHECK(hipMemset(d_spectrum_1, 0, d_proccnt * nstep_valid * nspec * sizeof(hipComplex)));

            CUDACHECK(hipMemset(d_timesignal_2, 0, d_proccnt * npts * sizeof(float)));
            CUDACHECK(hipMemset(d_spectrum_2, 0, d_proccnt * nstep_valid * nspec * sizeof(hipComplex)));

            CUDACHECK(hipMemset(d_timesignal_3, 0, d_proccnt * npts * sizeof(float)));
            CUDACHECK(hipMemset(d_spectrum_3, 0, d_proccnt * nstep_valid * nspec * sizeof(hipComplex)));

            // Read SAC Data
            CUDACHECK(hipMemcpy2D(d_timesignal_1, npts * sizeof(float),
                                   h_timesignal_1 + d_finishcnt * npts, npts * sizeof(float),
                                   npts * sizeof(float), d_proccnt, hipMemcpyHostToDevice));

            CUDACHECK(hipMemcpy2D(d_timesignal_2, npts * sizeof(float),
                                   h_timesignal_2 + d_finishcnt * npts, npts * sizeof(float),
                                   npts * sizeof(float), d_proccnt, hipMemcpyHostToDevice));

            CUDACHECK(hipMemcpy2D(d_timesignal_3, npts * sizeof(float),
                                   h_timesignal_3 + d_finishcnt * npts, npts * sizeof(float),
                                   npts * sizeof(float), d_proccnt, hipMemcpyHostToDevice));

            for (int stepidx = 0, done_step_flag = 0; stepidx < nstep; stepidx++)
            {
                // skip any step?
                if (skip_step == stepidx)
                    continue;

                // clean d_segment_timesignal_* and d_segment_spectrum_*
                CUDACHECK(hipMemset(d_segment_timesignal_1, 0, d_proccnt * nfft_2x * sizeof(float)));
                CUDACHECK(hipMemset(d_segment_spectrum_1, 0, d_proccnt * nfft_2x * sizeof(hipComplex)));

                CUDACHECK(hipMemset(d_segment_timesignal_2, 0, d_proccnt * nfft_2x * sizeof(float)));
                CUDACHECK(hipMemset(d_segment_spectrum_2, 0, d_proccnt * nfft_2x * sizeof(hipComplex)));

                CUDACHECK(hipMemset(d_segment_timesignal_3, 0, d_proccnt * nfft_2x * sizeof(float)));
                CUDACHECK(hipMemset(d_segment_spectrum_3, 0, d_proccnt * nfft_2x * sizeof(hipComplex)));

                // copy d_timesignal_* to d_segment_timesignal_*
                CUDACHECK(hipMemcpy2D(d_segment_timesignal_1, nfft_2x * sizeof(float),
                                       d_timesignal_1 + stepidx * nseg, npts * sizeof(float),
                                       nseg * sizeof(float), d_proccnt, hipMemcpyDeviceToDevice));
                CUDACHECK(hipMemcpy2D(d_segment_timesignal_2, nfft_2x * sizeof(float),
                                       d_timesignal_2 + stepidx * nseg, npts * sizeof(float),
                                       nseg * sizeof(float), d_proccnt, hipMemcpyDeviceToDevice));
                CUDACHECK(hipMemcpy2D(d_segment_timesignal_3, nfft_2x * sizeof(float),
                                       d_timesignal_3 + stepidx * nseg, npts * sizeof(float),
                                       nseg * sizeof(float), d_proccnt, hipMemcpyDeviceToDevice));

                // pre-processing of d_segment_timesignal_*
                preprocess(d_segment_timesignal_1, d_sum, d_isum, nseg, nfft_2x, d_proccnt, TAPER_RATIO);
                preprocess(d_segment_timesignal_2, d_sum, d_isum, nseg, nfft_2x, d_proccnt, TAPER_RATIO);
                preprocess(d_segment_timesignal_3, d_sum, d_isum, nseg, nfft_2x, d_proccnt, TAPER_RATIO);

                if (wh_before || do_runabs)
                {
                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_1, (hipfftComplex *)d_segment_spectrum_1));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_1, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_2, (hipfftComplex *)d_segment_spectrum_2));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_2, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_3, (hipfftComplex *)d_segment_spectrum_3));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_3, nfft_2x, nfft_2x, d_proccnt, delta);

                    if (wh_before)
                    {
                        freqWhiten_c9(d_segment_spectrum_1, d_segment_spectrum_2, d_segment_spectrum_3,
                                      d_weight_1, d_weight_2, d_weight_3,
                                      d_tmp, nfft_2x, nspec, d_proccnt, nsmooth, df, freq_low_limit, freq_high_limit, FILTERFLAG);
                    }
                    if (do_onebit)
                    {
                        CUFFTCHECK(hipfftExecC2R(planinv, (hipfftComplex *)d_segment_spectrum_1, (hipfftReal *)d_segment_timesignal_1));
                        InvNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_timesignal_1, nfft_2x, nfft_2x, d_proccnt, delta);

                        CUFFTCHECK(hipfftExecC2R(planinv, (hipfftComplex *)d_segment_spectrum_2, (hipfftReal *)d_segment_timesignal_2));
                        InvNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_timesignal_2, nfft_2x, nfft_2x, d_proccnt, delta);

                        CUFFTCHECK(hipfftExecC2R(planinv, (hipfftComplex *)d_segment_spectrum_3, (hipfftReal *)d_segment_timesignal_3));
                        InvNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_timesignal_3, nfft_2x, nfft_2x, d_proccnt, delta);
                    }
                }
                if (do_runabs)
                {
                    runabs_c9(d_segment_timesignal_1,
                              d_segment_timesignal_2,
                              d_segment_timesignal_3,
                              d_segment_spectrum_1,
                              d_segment_spectrum_2,
                              d_segment_spectrum_3,
                              d_filtered_segment_timesignal_1,
                              d_filtered_segment_timesignal_2,
                              d_filtered_segment_timesignal_3,
                              d_filtered_segment_spectrum_1,
                              d_filtered_segment_spectrum_2,
                              d_filtered_segment_spectrum_3,
                              d_weight_1, d_weight_2, d_weight_3,
                              d_tmp,
                              &planinv,
                              d_filter_responses,
                              freq_lows,
                              filter_count, delta, d_proccnt, nfft_2x,
                              df, MAXVAL);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_1, (hipfftComplex *)d_segment_spectrum_1));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_1, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_2, (hipfftComplex *)d_segment_spectrum_2));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_2, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_3, (hipfftComplex *)d_segment_spectrum_3));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_3, nfft_2x, nfft_2x, d_proccnt, delta);
                }

                if (do_onebit)
                {
                    onebit(d_segment_timesignal_1, nfft_2x, d_proccnt);
                    onebit(d_segment_timesignal_2, nfft_2x, d_proccnt);
                    onebit(d_segment_timesignal_3, nfft_2x, d_proccnt);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_1, (hipfftComplex *)d_segment_spectrum_1));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_1, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_2, (hipfftComplex *)d_segment_spectrum_2));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_2, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_3, (hipfftComplex *)d_segment_spectrum_3));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_3, nfft_2x, nfft_2x, d_proccnt, delta);
                }

                if (!do_runabs && !do_onebit && !wh_before)
                {
                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_1, (hipfftComplex *)d_segment_spectrum_1));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_1, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_2, (hipfftComplex *)d_segment_spectrum_2));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_2, nfft_2x, nfft_2x, d_proccnt, delta);

                    CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_segment_timesignal_3, (hipfftComplex *)d_segment_spectrum_3));
                    FwdNormalize2DKernel<<<dimgrd, dimblk>>>(d_segment_spectrum_3, nfft_2x, nfft_2x, d_proccnt, delta);
                }

                if (wh_after)
                {
                    freqWhiten_c9(d_segment_spectrum_1, d_segment_spectrum_2, d_segment_spectrum_3,
                                  d_weight_1, d_weight_2, d_weight_3,
                                  d_tmp, nfft_2x, nspec, d_proccnt, nsmooth, df,
                                  freq_low_limit, freq_high_limit, FILTERFLAG);
                }

                CUDACHECK(hipMemcpy2D(d_spectrum_1 + done_step_flag * nspec, nstep_valid * nspec * sizeof(hipComplex),
                                       d_segment_spectrum_1, nfft_2x * sizeof(hipComplex),
                                       nspec * sizeof(hipComplex), d_proccnt, hipMemcpyDeviceToDevice));

                CUDACHECK(hipMemcpy2D(d_spectrum_2 + done_step_flag * nspec, nstep_valid * nspec * sizeof(hipComplex),
                                       d_segment_spectrum_2, nfft_2x * sizeof(hipComplex),
                                       nspec * sizeof(hipComplex), d_proccnt, hipMemcpyDeviceToDevice));

                CUDACHECK(hipMemcpy2D(d_spectrum_3 + done_step_flag * nspec, nstep_valid * nspec * sizeof(hipComplex),
                                       d_segment_spectrum_3, nfft_2x * sizeof(hipComplex),
                                       nspec * sizeof(hipComplex), d_proccnt, hipMemcpyDeviceToDevice));
                done_step_flag++;
            } // End loop of each segment
            CUDACHECK(hipMemcpy2D(h_spectrum_1 + d_finishcnt * nstep_valid * nspec,
                                   nstep_valid * nspec * sizeof(hipComplex), d_spectrum_1,
                                   nstep_valid * nspec * sizeof(hipComplex),
                                   nstep_valid * nspec * sizeof(hipComplex), d_proccnt,
                                   hipMemcpyDeviceToHost));
            CUDACHECK(hipMemcpy2D(h_spectrum_2 + d_finishcnt * nstep_valid * nspec,
                                   nstep_valid * nspec * sizeof(hipComplex), d_spectrum_2,
                                   nstep_valid * nspec * sizeof(hipComplex),
                                   nstep_valid * nspec * sizeof(hipComplex), d_proccnt,
                                   hipMemcpyDeviceToHost));
            CUDACHECK(hipMemcpy2D(h_spectrum_3 + d_finishcnt * nstep_valid * nspec,
                                   nstep_valid * nspec * sizeof(hipComplex), d_spectrum_3,
                                   nstep_valid * nspec * sizeof(hipComplex),
                                   nstep_valid * nspec * sizeof(hipComplex), d_proccnt,
                                   hipMemcpyDeviceToHost));

        } // Quit GPU processing
        // Writing the output spectra
        parallel_write_spec_c9(write_pool, h_proccnt, pInOutList, MAX_THREADS);
        destroy_threadpool_write(write_pool);
        destroy_threadpool_read(read_pool);
    }
    // free memory
    freeMemory(planfwd, planinv,
               &d_timesignal_1, &d_spectrum_1,
               &d_timesignal_2, &d_spectrum_2,
               &d_timesignal_3, &d_spectrum_3,
               &d_segment_timesignal_1, &d_segment_spectrum_1,
               &d_segment_timesignal_2, &d_segment_spectrum_2,
               &d_segment_timesignal_3, &d_segment_spectrum_3,
               &d_filtered_segment_timesignal_1, &d_filtered_segment_spectrum_1,
               &d_filtered_segment_timesignal_2, &d_filtered_segment_spectrum_2,
               &d_filtered_segment_timesignal_3, &d_filtered_segment_spectrum_3,
               &d_filter_responses,
               &d_weight_1, &d_weight_2, &d_weight_3,
               &d_sum, &d_isum, &d_tmp,
               &h_timesignal_1, &h_spectrum_1,
               &h_timesignal_2, &h_spectrum_2,
               &h_timesignal_3, &h_spectrum_3,
               pInOutList);
}
