#include "hip/hip_runtime.h"
#include "cuda.misc.cuh"
#include <cstddef>

/* Define Kernel Fucntions */
__global__ void abs2DKernel(float *d_data, size_t pitch, size_t width,
                            size_t height)
{
  size_t col = blockIdx.x * blockDim.x + threadIdx.x;
  size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < width && row < height)
  {
    size_t idx = row * pitch + col;
    d_data[idx] = fabs(d_data[idx]);
  }
}

__global__ void clampmin2DKernel(float *d_data, size_t pitch, size_t width,
                                 size_t height, float minval)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int idx = row * pitch + col;
    if (d_data[idx] < minval)
    {
      d_data[idx] = minval;
    }
  }
}

__global__ void isnan2DKernel(float *d_data, size_t pitch, size_t width,
                              size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < width && row < height)
  {
    int idx = row * pitch + col;
    if (isnan(d_data[idx]) || isinf(d_data[idx]))
    {
      d_data[idx] = 0;
    }
  }
}

__global__ void cisnan2DKernel(hipComplex *d_data, size_t pitch, size_t width,
                               size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int idx = row * pitch + col;
    if (isnan(d_data[idx].x) || isinf(d_data[idx].x) || isnan(d_data[idx].y) ||
        isinf(d_data[idx].y))
    {
      d_data[idx].x = 0;
      d_data[idx].y = 0;
    }
  }
}

__global__ void div2DKernel(float *d_data, size_t dpitch, float *d_divisor,
                            size_t spitch, size_t width, size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int sidx = row * spitch + col;
    int didx = row * dpitch + col;
    d_data[didx] = d_data[sidx] / d_divisor[sidx];
  }
}

__global__ void cdiv2DKernel(hipComplex *d_data, size_t dpitch, float *d_divisor,
                             size_t spitch, size_t width, size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int sidx = row * spitch + col;
    int didx = row * dpitch + col;
    d_data[didx].x /= d_divisor[sidx];
    d_data[didx].y /= d_divisor[sidx];
  }
}

__global__ void sum2DKernel(float *d_data_out, size_t dpitch, float *d_data_in,
                            size_t spitch, size_t width, size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int sidx = row * spitch + col;
    int didx = row * dpitch + col;
    d_data_out[didx] = d_data_out[sidx] + d_data_in[sidx];
  }
}

__global__ void cutmax2DKernel(float *d_data, size_t pitch, size_t width,
                               size_t height, float maxval)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int idx = row * pitch + col;

    float val = d_data[idx];

    if (val > maxval)
    {
      d_data[idx] = maxval;
    }
    else if (val < -1 * maxval)
    {
      d_data[idx] = -1 * maxval;
    }
  }
}

__global__ void amp2DKernel(float *d_amp, size_t dpitch, hipComplex *d_data,
                            size_t spitch, size_t width, size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    if (col == 0)
    {
      d_amp[row * dpitch] = fabs(hipCrealf(d_data[row * spitch]));
      d_amp[row * dpitch + width] = fabs(hipCimagf(d_data[row * spitch]));
    }
    else
    {
      hipComplex c = d_data[row * spitch + col];
      d_amp[row * dpitch + col] = hipCabsf(c);
    }
  }
}

__global__ void filterKernel(hipComplex *d_spectrum, hipComplex *d_response, size_t pitch, size_t width, size_t height)
{
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height)
  {
    int idx = row * pitch + col;
    // filter the first time
    d_spectrum[idx] = hipCmulf(d_spectrum[idx], d_response[col]);

    // doing conjugate, to reverse the time direction
    d_spectrum[idx] = hipConjf(d_spectrum[idx]);

    // filter the second time
    d_spectrum[idx] = hipCmulf(d_spectrum[idx], d_response[col]);

    // doing conjugate, to reverse the time direction again
    d_spectrum[idx] = hipConjf(d_spectrum[idx]);
  }
}
