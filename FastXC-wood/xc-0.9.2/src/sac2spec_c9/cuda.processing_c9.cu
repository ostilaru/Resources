#include "hip/hip_runtime.h"
#include "cuda.processing_c9.cuh"

// pre-processing for sacdat: isnan, demean, detrend
void preprocess(float *d_sacdata, double *d_sum, double *d_isum, int npts,
                int pitch, size_t proccnt, int taper_percentage)
{
    // check the nan/inf value of [d_sacdata]
    size_t width = npts;
    size_t height = proccnt;
    dim3 dimgrd, dimblk;
    DimCompute(&dimgrd, &dimblk, width, height);

    dim3 dimgrd2, dimblk2;
    dimblk2.x = BLOCKMAX;
    dimblk2.y = 1;
    dimgrd2.x = 1;
    dimgrd2.y = height;

    isnan2DKernel<<<dimgrd, dimblk>>>(d_sacdata, pitch, width, height);

    // demean. First calculate the mean value of each trace

    size_t dpitch = 1;
    size_t spitch = pitch;
    sumSingleBlock2DKernel<<<dimgrd2, dimblk2,
                             dimblk2.x * dimblk2.y * sizeof(double)>>>(
        d_sum, dpitch, d_sacdata, spitch, width, height);

    DimCompute(&dimgrd, &dimblk, width, height);
    rdc2DKernel<<<dimgrd, dimblk>>>(d_sacdata, pitch, width, height,
                                    d_sum);

    // detrend. First calculate d_sum and d_isum

    sumSingleBlock2DKernel<<<dimgrd2, dimblk2,
                             dimblk2.x * dimblk2.y * sizeof(double)>>>(
        d_sum, dpitch, d_sacdata, spitch, width, height);

    isumSingleBlock2DKernel<<<dimgrd2, dimblk2,
                              dimblk2.x * dimblk2.y * sizeof(double)>>>(
        d_isum, dpitch, d_sacdata, spitch, width, height);

    rtr2DKernel<<<dimgrd, dimblk>>>(d_sacdata, pitch, width, height,
                                    d_sum, d_isum);

    // taper, taper percentage set in config.h
    timetaper2DKernel<<<dimgrd, dimblk>>>(d_sacdata, pitch, width, height, taper_percentage);
}

/* multi-frequency time domain normalization */
void runabs_onefreq_c9(hipComplex *d_filtered_spectrum_1,
                       hipComplex *d_filtered_spectrum_2,
                       hipComplex *d_filtered_spectrum_3,
                       float *d_filtered_timesignal_1,
                       float *d_filtered_timesignal_2,
                       float *d_filtered_timesignal_3,
                       hipComplex *d_response,
                       float *d_weight_1,
                       float *d_weight_2,
                       float *d_weight_3,
                       float *d_tmp,
                       int nhalf_average_win,
                       float delta, int proccnt, int nfft_1x, float maxval,
                       hipfftHandle *planinv)
{
    // set the width and height of data
    size_t twidth = nfft_1x;         // time domain width
    size_t fwidth = nfft_1x / 2 + 1; // frequency domain width
    size_t height = proccnt;
    size_t pitch = nfft_1x;

    // calculate the grid and block size for time domain and frequency domain
    dim3 tdimgrd, tdimblk;
    DimCompute(&tdimgrd, &tdimblk, twidth, height);

    dim3 fdimgrd, fdimblk;
    DimCompute(&fdimgrd, &fdimblk, fwidth, height);

    filterKernel<<<fdimgrd, fdimblk>>>(d_filtered_spectrum_1, d_response, pitch, fwidth, height);
    filterKernel<<<fdimgrd, fdimblk>>>(d_filtered_spectrum_2, d_response, pitch, fwidth, height);
    filterKernel<<<fdimgrd, fdimblk>>>(d_filtered_spectrum_3, d_response, pitch, fwidth, height);

    CUFFTCHECK(hipfftExecC2R(*planinv, (hipfftComplex *)d_filtered_spectrum_1, (hipfftReal *)d_filtered_timesignal_1));
    InvNormalize2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_1, pitch, twidth, height, delta);

    CUFFTCHECK(hipfftExecC2R(*planinv, (hipfftComplex *)d_filtered_spectrum_2, (hipfftReal *)d_filtered_timesignal_2));
    InvNormalize2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_2, pitch, twidth, height, delta);

    CUFFTCHECK(hipfftExecC2R(*planinv, (hipfftComplex *)d_filtered_spectrum_3, (hipfftReal *)d_filtered_timesignal_3));
    InvNormalize2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_3, pitch, twidth, height, delta);

    CUDACHECK(hipMemcpy2D(d_weight_1, nfft_1x * sizeof(float),
                           d_filtered_timesignal_1, nfft_1x * sizeof(float),
                           nfft_1x * sizeof(float), proccnt, hipMemcpyDeviceToDevice));

    CUDACHECK(hipMemcpy2D(d_weight_2, nfft_1x * sizeof(float),
                           d_filtered_timesignal_2, nfft_1x * sizeof(float),
                           nfft_1x * sizeof(float), proccnt, hipMemcpyDeviceToDevice));

    CUDACHECK(hipMemcpy2D(d_weight_3, nfft_1x * sizeof(float),
                           d_filtered_timesignal_3, nfft_1x * sizeof(float),
                           nfft_1x * sizeof(float), proccnt, hipMemcpyDeviceToDevice));

    abs2DKernel<<<tdimgrd, tdimblk>>>(d_weight_1, pitch, twidth, height);
    abs2DKernel<<<tdimgrd, tdimblk>>>(d_weight_2, pitch, twidth, height);
    abs2DKernel<<<tdimgrd, tdimblk>>>(d_weight_3, pitch, twidth, height);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_1, pitch * sizeof(float), twidth * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothTime2DKernel<<<tdimgrd, tdimblk>>>(d_weight_1, pitch, d_tmp, pitch, twidth, height, nhalf_average_win);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_2, pitch * sizeof(float), twidth * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothTime2DKernel<<<tdimgrd, tdimblk>>>(d_weight_2, pitch, d_tmp, pitch, twidth, height, nhalf_average_win);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_3, pitch * sizeof(float), twidth * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothTime2DKernel<<<tdimgrd, tdimblk>>>(d_weight_3, pitch, d_tmp, pitch, twidth, height, nhalf_average_win);

    // add to d_weight_1 as the final weight
    sum2DKernel<<<tdimgrd, tdimblk>>>(d_weight_1, pitch, d_weight_2, pitch, twidth, height);
    sum2DKernel<<<tdimgrd, tdimblk>>>(d_weight_1, pitch, d_weight_3, pitch, twidth, height);

    // Avoid the minimum value is zero, old version is cunzero2D
    clampmin2DKernel<<<tdimgrd, tdimblk>>>(d_weight_1, pitch, twidth, height, MINVAL);

    div2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_1, pitch, d_weight_1, pitch, twidth, height);
    div2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_2, pitch, d_weight_1, pitch, twidth, height);
    div2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_3, pitch, d_weight_1, pitch, twidth, height);

    isnan2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_1, pitch, twidth, height);
    isnan2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_2, pitch, twidth, height);
    isnan2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_3, pitch, twidth, height);

    cutmax2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_1, pitch, twidth, height, maxval);
    cutmax2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_2, pitch, twidth, height, maxval);
    cutmax2DKernel<<<tdimgrd, tdimblk>>>(d_filtered_timesignal_3, pitch, twidth, height, maxval);
}

// multi-frequency time domain normalization
void runabs_c9(float *d_timesignal_1,
               float *d_timesignal_2,
               float *d_timesignal_3,
               hipComplex *d_spectrum_1,
               hipComplex *d_spectrum_2,
               hipComplex *d_spectrum_3,
               float *d_filtered_timesignal_1,
               float *d_filtered_timesignal_2,
               float *d_filtered_timesignal_3,
               hipComplex *d_filtered_spectrum_1,
               hipComplex *d_filtered_spectrum_2,
               hipComplex *d_filtered_spectrum_3,
               float *d_weight_1, float *d_weight_2, float *d_weight_3,
               float *d_tmp,
               hipfftHandle *planinv,
               hipComplex *d_filter_responses,
               float *freq_lows,
               int filterCount, float delta, int proccnt,
               int nfft_1x, float maxval)
{
    size_t twidth = nfft_1x;
    size_t fwidth = nfft_1x * 0.5 + 1;
    size_t height = proccnt;
    size_t pitch = nfft_1x;

    // calculate the grid and block size for time domain and frequency domain
    dim3 dimgrd, dimblk;
    DimCompute(&dimgrd, &dimblk, twidth, height);

    // set d_timesignal_* to zero for the output
    CUDACHECK(hipMemset(d_timesignal_1, 0, proccnt * nfft_1x * sizeof(float)));
    CUDACHECK(hipMemset(d_timesignal_2, 0, proccnt * nfft_1x * sizeof(float)));
    CUDACHECK(hipMemset(d_timesignal_3, 0, proccnt * nfft_1x * sizeof(float)));

    hipComplex *d_response = NULL;
    hipMalloc((void **)&d_response, nfft_1x * sizeof(hipComplex));
    hipMemset(d_response, 0, nfft_1x * sizeof(hipComplex));

    // time domain normalization on different frequency and add them together
    for (int i = 0; i < filterCount; i++)
    {
        // get the current filter response
        CUDACHECK(hipMemcpy2D(d_response, nfft_1x * sizeof(hipComplex),
                               d_filter_responses + i * nfft_1x, nfft_1x * sizeof(hipComplex),
                               fwidth * sizeof(hipComplex), 1, hipMemcpyDeviceToDevice));

        // refrence from Yao's code winsize = SampleF * EndT
        int nhalf_average_win = int(1.0 / (freq_lows[i] * delta)) + 1;

        CUDACHECK(hipMemcpy2D(d_filtered_spectrum_1, nfft_1x * sizeof(hipComplex),
                               d_spectrum_1, nfft_1x * sizeof(hipComplex),
                               fwidth * sizeof(hipComplex), proccnt, hipMemcpyDeviceToDevice));

        CUDACHECK(hipMemcpy2D(d_filtered_spectrum_2, nfft_1x * sizeof(hipComplex),
                               d_spectrum_2, nfft_1x * sizeof(hipComplex),
                               fwidth * sizeof(hipComplex), proccnt, hipMemcpyDeviceToDevice));

        CUDACHECK(hipMemcpy2D(d_filtered_spectrum_3, nfft_1x * sizeof(hipComplex),
                               d_spectrum_3, nfft_1x * sizeof(hipComplex),
                               fwidth * sizeof(hipComplex), proccnt, hipMemcpyDeviceToDevice));

        CUDACHECK(hipMemset(d_filtered_timesignal_1, 0, proccnt * nfft_1x * sizeof(float)));
        CUDACHECK(hipMemset(d_filtered_timesignal_2, 0, proccnt * nfft_1x * sizeof(float)));
        CUDACHECK(hipMemset(d_filtered_timesignal_3, 0, proccnt * nfft_1x * sizeof(float)));

        CUDACHECK(hipMemset(d_weight_1, 0, proccnt * nfft_1x * sizeof(float)));
        CUDACHECK(hipMemset(d_weight_2, 0, proccnt * nfft_1x * sizeof(float)));
        CUDACHECK(hipMemset(d_weight_3, 0, proccnt * nfft_1x * sizeof(float)));

        CUDACHECK(hipMemset(d_tmp, 0, proccnt * nfft_1x * sizeof(float)));

        runabs_onefreq_c9(d_filtered_spectrum_1,
                          d_filtered_spectrum_2,
                          d_filtered_spectrum_3,
                          d_filtered_timesignal_1,
                          d_filtered_timesignal_2,
                          d_filtered_timesignal_3,
                          d_response,
                          d_weight_1, d_weight_2, d_weight_3,
                          d_tmp,
                          nhalf_average_win,
                          delta, proccnt, nfft_1x,
                          maxval, planinv);

        dim3 tdimgrd, tdimblk;
        DimCompute(&tdimgrd, &tdimblk, twidth, height);

        // add different freq result [d_segdata_filted] to output [d_dataseg]
        sum2DKernel<<<dimgrd, dimblk>>>(d_timesignal_1, pitch, d_filtered_timesignal_1, pitch, twidth, height);
        sum2DKernel<<<dimgrd, dimblk>>>(d_timesignal_2, pitch, d_filtered_timesignal_2, pitch, twidth, height);
        sum2DKernel<<<dimgrd, dimblk>>>(d_timesignal_3, pitch, d_filtered_timesignal_3, pitch, twidth, height);
    }
}

void freqWhiten_c9(hipComplex *d_spectrum_1,
                   hipComplex *d_spectrum_2,
                   hipComplex *d_spectrum_3,
                   float *d_weight_1, float *d_weight_2, float *d_weight_3,
                   float *d_tmp, int pitch, int width,
                   int height, int nsmooth, float df, float freq_low_limit, float freq_high_limit)
{
    dim3 dimgrd, dimblk;
    DimCompute(&dimgrd, &dimblk, width, height);

    amp2DKernel<<<dimgrd, dimblk>>>(d_weight_1, pitch, d_spectrum_1, pitch, width, height);
    amp2DKernel<<<dimgrd, dimblk>>>(d_weight_2, pitch, d_spectrum_2, pitch, width, height);
    amp2DKernel<<<dimgrd, dimblk>>>(d_weight_3, pitch, d_spectrum_3, pitch, width, height);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_1, pitch * sizeof(float),
                           width * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothFreq2DKernel<<<dimgrd, dimblk>>>(d_weight_1, pitch, d_tmp, pitch, width, height, nsmooth);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_2, pitch * sizeof(float),
                           width * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothFreq2DKernel<<<dimgrd, dimblk>>>(d_weight_2, pitch, d_tmp, pitch, width, height, nsmooth);

    CUDACHECK(hipMemcpy2D(d_tmp, pitch * sizeof(float), d_weight_3, pitch * sizeof(float),
                           width * sizeof(float), height, hipMemcpyDeviceToDevice));
    smoothFreq2DKernel<<<dimgrd, dimblk>>>(d_weight_3, pitch, d_tmp, pitch, width, height, nsmooth);

    clampmin2DKernel<<<dimgrd, dimblk>>>(d_weight_1, pitch, width, height, MINVAL);
    clampmin2DKernel<<<dimgrd, dimblk>>>(d_weight_2, pitch, width, height, MINVAL);
    clampmin2DKernel<<<dimgrd, dimblk>>>(d_weight_3, pitch, width, height, MINVAL);

    // add to d_weight_1 as the final weight
    sum2DKernel<<<dimgrd, dimblk>>>(d_weight_1, pitch, d_weight_2, pitch, width, height);
    sum2DKernel<<<dimgrd, dimblk>>>(d_weight_1, pitch, d_weight_3, pitch, width, height);

    cdiv2DKernel<<<dimgrd, dimblk>>>(d_spectrum_1, pitch, d_weight_1, pitch, width, height);
    cdiv2DKernel<<<dimgrd, dimblk>>>(d_spectrum_2, pitch, d_weight_1, pitch, width, height);
    cdiv2DKernel<<<dimgrd, dimblk>>>(d_spectrum_3, pitch, d_weight_1, pitch, width, height);

    // Apply spectra taper on [d_segspec]
    int idx1 = int(freq_low_limit * 0.667 / df);
    int idx2 = int(freq_low_limit / df);
    int idx3 = int(freq_high_limit / df);
    int idx4 = int(freq_high_limit * 1.333 / df);

    int np = 1; // window order

    // Using cosine window
    specTaper2DCosineKernel<<<dimgrd, dimblk>>>(d_spectrum_1, pitch, width, height, np, idx1, idx2, idx3, idx4);
    specTaper2DCosineKernel<<<dimgrd, dimblk>>>(d_spectrum_2, pitch, width, height, np, idx1, idx2, idx3, idx4);
    specTaper2DCosineKernel<<<dimgrd, dimblk>>>(d_spectrum_3, pitch, width, height, np, idx1, idx2, idx3, idx4);
}
