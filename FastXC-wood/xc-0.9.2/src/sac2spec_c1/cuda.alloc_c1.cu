#include "cuda.alloc_c1.cuh"
size_t EstimateGpuBatchC1(size_t gpu_id, int npts, int nfft_1x, int nstep, int filter_count, size_t wh_flag, size_t runabs_flag)
{
    int nfft_2x = nfft_1x * 2;

    // CuFFT parameter
    int rank = 1;
    int n[1] = {nfft_1x};
    int inembed[1] = {nfft_1x};
    int onembed[1] = {nfft_1x};
    int istride = 1;
    int idist = nfft_1x;
    int ostride = 1;
    int odist = nfft_1x;

    // CuFFT parameter for 2x zero padding data
    int rank_2x = 1;
    int n_2x[1] = {nfft_2x};
    int inembed_2x[1] = {nfft_2x};
    int onembed_2x[1] = {nfft_2x};
    int istride_2x = 1;
    int idist_2x = nfft_2x;
    int ostride_2x = 1;
    int odist_2x = nfft_2x;

    // unitgpuram setting
    size_t sac_size = npts * sizeof(float);                         // d_sacdata
    size_t spec_size = nstep * (nfft_2x + 1) * sizeof(hipComplex);   // d_spectrum
    size_t sac_seg_size = nfft_1x * sizeof(float);                  // d_segsac
    size_t spec_seg_size = nfft_1x * sizeof(hipComplex);             // d_segspec
    size_t sac_seg_2x_size = nfft_2x * sizeof(float);               // d_segsac_2x with zero padding
    size_t spec_seg_2x_size = nfft_2x * sizeof(hipComplex);          // d_segspec_2x with zero padding

    // gpuram for preprocessing rdc and rtr
    size_t pre_process_size = sizeof(double)    // d_sum
                              + sizeof(double); // d_isum

    size_t fixed_size = nfft_1x * sizeof(hipComplex) * (filter_count + 1); // fixed memory for filter response

    // calcaulate gpuram for frequency whiten and time normalization
    size_t whiten_norm_size = 0;
    if (runabs_flag)
    {                                                    // If runabs normalization is applied
        whiten_norm_size = nfft_1x * sizeof(float)       // d_weight
                           + nfft_1x * sizeof(float)     // d_segsac_tmp
                           + nfft_1x * sizeof(hipComplex) // d_segspec_tmp
                           + nfft_1x * sizeof(double);   // d_tmp
    }
    else if (!runabs_flag && wh_flag)
    {                                                 // If only frequency whiten is applied
        whiten_norm_size = nfft_1x * sizeof(float)    // d_weight
                           + nfft_1x * sizeof(float); // d_tmp
    }

    size_t unitgpuram =
        sac_size            // input sac data npts * float
        + spec_size         // output spectrum data nstep * nspec * hipComplex
        + sac_seg_size      // segment sac data nfft * float (nfft is redundant)
        + spec_seg_size     // segment spectrum data nfft * hipComplex
        + sac_seg_2x_size   // zero-padding segment spectrum data nfft * float
        + spec_seg_2x_size  // zero-padding segment spectrum data nfft * hipComplex
        + pre_process_size  // d_sum, d_isum
        + whiten_norm_size; // whiten and normalization

    size_t availram = QueryAvailGpuRam(gpu_id);
    size_t reqram = 0;
    size_t tmpram = 0;
    size_t batch = 0;
    while (true)
    {
        batch++;
        reqram = fixed_size + batch * unitgpuram;
        // cuFFT memory usage for data fft forward
        hipfftEstimateMany(rank, n, inembed, istride, idist, onembed, ostride,
                          odist, HIPFFT_R2C, batch, &tmpram);
        reqram += tmpram;
        // cuFFT memory usage for data fft inverse
        hipfftEstimateMany(rank, n, inembed, istride, idist, onembed, ostride,
                          odist, HIPFFT_C2R, batch, &tmpram);
        
        reqram += tmpram;
        
        // cuFFT memory usage for zero padding data fft forward
        hipfftEstimateMany(rank_2x, n_2x, inembed_2x, istride_2x, idist_2x, onembed_2x, ostride_2x,
                          odist_2x, HIPFFT_R2C, batch, &tmpram);

        reqram += tmpram;

        if (reqram > availram)
        {          // Check if reqram exceeds availram
            break; // Exit the loop
        }
    }
    batch = batch > _RISTRICT_MAX_GPU_BATCH ? _RISTRICT_MAX_GPU_BATCH : batch;
    return batch;
}

void allocateCudaMemoryC1(int d_batch, int npts, int nstep_valid, int nfft_1x,
                          int do_runabs, int wh_flag,
                          float **d_timesignal,
                          hipComplex **d_spectrum,
                          float **d_segment_timesignal,
                          hipComplex **d_segment_spectrum,
                          float **d_segment_timesignal_2x,
                          hipComplex **d_segment_spectrum_2x,
                          float **d_filtered_segment_timesignal,
                          hipComplex **d_filtered_segment_spectrum,
                          hipComplex **d_filter_responses,
                          int filterCount,
                          float **d_weight, float **d_tmp,
                          double **d_sum, double **d_isum,
                          hipfftHandle *planfwd, hipfftHandle *planinv, hipfftHandle *planfwd_2x)
{
    // Variables for input and output
    int nfft_2x = nfft_1x * 2;
    hipMalloc((void **)d_timesignal, d_batch * npts * sizeof(float));
    hipMalloc((void **)d_spectrum, d_batch * nstep_valid * (nfft_2x / 2 + 1) * sizeof(hipComplex));

    // Variables for processing segment data
    hipMalloc((void **)d_segment_timesignal, d_batch * nfft_1x * sizeof(float));
    hipMalloc((void **)d_segment_spectrum, d_batch * nfft_1x * sizeof(hipComplex));
    hipMalloc((void **)d_segment_timesignal_2x, d_batch * nfft_2x * sizeof(float));
    hipMalloc((void **)d_segment_spectrum_2x, d_batch * nfft_2x * sizeof(hipComplex));

    hipMalloc((void **)d_filter_responses, filterCount * nfft_1x * sizeof(hipComplex));
    hipMalloc((void **)d_sum, d_batch * sizeof(double));
    hipMalloc((void **)d_isum, d_batch * sizeof(double));

    // if whiten, allocate memory for weight and tmp, same size as segment data
    if (!do_runabs && wh_flag)
    {
        hipMalloc((void **)d_weight, d_batch * nfft_1x * sizeof(float));
        hipMalloc((void **)d_tmp, d_batch * nfft_1x * sizeof(float));
    } // if runabs, allocate memory for filterd sac and spec
    else if (do_runabs)
    {
        hipMalloc((void **)d_filtered_segment_timesignal, d_batch * nfft_1x * sizeof(float));
        hipMalloc((void **)d_filtered_segment_spectrum, d_batch * nfft_1x * sizeof(hipComplex));
        hipMalloc((void **)d_weight, d_batch * nfft_1x * sizeof(float));
        hipMalloc((void **)d_tmp, d_batch * nfft_1x * sizeof(float));
    }

    // set up cufft plans
    int rank = 1;
    int n[1] = {nfft_1x};
    int inembed[1] = {nfft_1x};
    int onembed[1] = {nfft_1x};
    int istride = 1;
    int idist = nfft_1x;
    int ostride = 1;
    int odist = nfft_1x;

    hipfftPlanMany(planfwd, rank, n, inembed, istride, idist, onembed,
                  ostride, odist, HIPFFT_R2C, d_batch);
    hipfftPlanMany(planinv, rank, n, inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2R, d_batch);
    
    // set up cufft plans for zero-padding
    int rank_2x = 1;
    int n_2x[1] = {nfft_2x};
    int inembed_2x[1] = {nfft_2x};
    int onembed_2x[1] = {nfft_2x};
    int istride_2x = 1;
    int idist_2x = nfft_2x;
    int ostride_2x = 1;
    int odist_2x = nfft_2x;

    hipfftPlanMany(planfwd_2x, rank_2x, n_2x, inembed_2x, istride_2x, idist_2x, onembed_2x,
                  ostride_2x, odist_2x, HIPFFT_R2C, d_batch);
}

// Free memory for the PathNode linked list
void free_PathList(PathNode *head)
{
    PathNode *current = head;
    PathNode *next_node;

    while (current != NULL)
    {
        next_node = current->next;

        // Free memory for the string
        if (current->path != NULL)
        {
            CpuFree((void **)&current->path);
        }

        // Free memory for the struct itself
        CpuFree((void **)&current);

        current = next_node;
    }
}

void freeMemory(hipfftHandle planfwd, hipfftHandle planinv,
                float **d_timesignal,
                hipComplex **d_spectrum,
                float **d_segment_timesignal,
                hipComplex **d_segment_spectrum,
                float **d_segment_timesignal_2x,
                hipComplex **d_segment_spectrum_2x,
                float **d_filtered_segment_timesignal,
                hipComplex **d_filtered_segment_spectrum,
                hipComplex **d_filter_responses,
                float **d_weight, float **d_tmp,
                double **d_sum, double **d_isum,
                float **h_timesignal, complex **h_spectrum,
                InOutNodeC1 *pInOutList)
{
    hipfftDestroy(planfwd);
    hipfftDestroy(planinv);

    GpuFree((void **)d_timesignal);
    GpuFree((void **)d_spectrum);

    GpuFree((void **)d_segment_timesignal);
    GpuFree((void **)d_segment_spectrum);

    GpuFree((void **)d_segment_timesignal_2x);
    GpuFree((void **)d_segment_spectrum_2x);

    GpuFree((void **)d_filtered_segment_timesignal);
    GpuFree((void **)d_filtered_segment_spectrum);
    GpuFree((void **)d_filter_responses);

    GpuFree((void **)d_weight);
    GpuFree((void **)d_tmp);

    GpuFree((void **)d_sum);
    GpuFree((void **)d_isum);

    CpuFree((void **)h_timesignal);
    CpuFree((void **)h_spectrum);
    CpuFree((void **)pInOutList);
}
