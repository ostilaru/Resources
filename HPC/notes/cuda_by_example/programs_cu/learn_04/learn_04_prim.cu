#include<iostream>
#include<hip/hip_runtime.h>
#include<cstdint>

const int matrixSize = 1024 * 40;

// __global__ void matrixMultiplication(uint8_t *a, uint8_t *b, uint8_t *c) {
//     int tx = threadIdx.x;
//     int ty = threadIdx.y;
//     int bx = blockIdx.x;
//     int by = blockIdx.y;

//     int row = by * blockDim.y + ty;
//     int col = bx * blockDim.x + tx;

//     __shared__ uint8_t shared_a[32][32];
//     __shared__ uint8_t shared_b[32][32];

//     int sum = 0;

//     for (int i = 0; i < matrixSize / 32; ++i) {
//         shared_a[ty][tx] = a[row * matrixSize + i * 32 + tx];
//         shared_b[ty][tx] = b[(i * 32 + ty) * matrixSize + col];
//         __syncthreads();

//         for (int k = 0; k < 32; ++k) {
//             sum += shared_a[ty][k] * shared_b[k][tx];
//         }
//         __syncthreads();
//     }

//     if (row < matrixSize && col < matrixSize) {
//         c[row * matrixSize + col] = static_cast<uint8_t>(sum);
//     }
// }

__global__ void matrixMultiplication(uint8_t *a, uint8_t *b, uint8_t *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < matrixSize && col < matrixSize) {
        uint8_t sum = 0;
        for (int k = 0; k < matrixSize; ++k) {
            sum += a[row * matrixSize + k] * b[k * matrixSize + col];
        }
        c[row * matrixSize + col] = sum;
    }
}

int main() {
    uint8_t *host_a = new uint8_t[matrixSize * matrixSize];
    uint8_t *host_b = new uint8_t[matrixSize * matrixSize];
    uint8_t *host_c = new uint8_t[matrixSize * matrixSize];

    // 初始化矩阵
    for (int i = 0; i < matrixSize * matrixSize; ++i) {
        host_a[i] = static_cast<uint8_t>(1);
        host_b[i] = static_cast<uint8_t>(2);
    }

    // 创建 CUDA 事件对象用于计时
    hipEvent_t startTotal, stopTotal, startGPU, stopGPU;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    // 记录开始时间（总的开销）
    hipEventRecord(startTotal);

    // ---------------------------------------------------------------
    // 在GPU上分配内存
    uint8_t *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, matrixSize * matrixSize * sizeof(uint8_t));
    hipMalloc((void**)&dev_b, matrixSize * matrixSize * sizeof(uint8_t));
    hipMalloc((void**)&dev_c, matrixSize * matrixSize * sizeof(uint8_t));

    // 将数据从device复制到GPU
    hipMemcpy(dev_a, host_a, matrixSize * matrixSize * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, matrixSize * matrixSize * sizeof(uint8_t), hipMemcpyHostToDevice);

    // 定义线程块和网格
    dim3 gridDim(matrixSize / 32, matrixSize / 32);
    dim3 blockDim(32, 32);

    // 记录开始时间（仅GPU计算）
    hipEventRecord(startGPU);

    // 调用GPU上的核函数
    matrixMultiplication<<<gridDim, blockDim>>>(dev_a, dev_b, dev_c);

    // 在主机上等待GPU计算完成
    hipDeviceSynchronize();

    // 记录结束时间（仅GPU计算）
    hipEventRecord(stopGPU);
    hipEventSynchronize(stopGPU);

    // 将结果从GPU复制回device
    hipMemcpy(host_c, dev_c, matrixSize * matrixSize * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // ---------------------------------------------------------------
    // 记录结束时间（总的开销）
    hipEventRecord(stopTotal);
    hipEventSynchronize(stopTotal);

    // 计算执行时间（总的开销）
    float millisecondsTotal = 0;
    hipEventElapsedTime(&millisecondsTotal, startTotal, stopTotal);

    // 计算执行时间（仅GPU计算）
    float millisecondsGPU = 0;
    hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);

    std::cout << "Total time: " << millisecondsTotal << " milliseconds" << std::endl;
    std::cout << "GPU calculation time: " << millisecondsGPU << " milliseconds" << std::endl;

    // 释放分配的内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    delete[] host_a;
    delete[] host_b;
    delete[] host_c;

    return 0;
}
